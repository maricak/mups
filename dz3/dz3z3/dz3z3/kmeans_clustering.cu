#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include "kmeans.h"

#define RANDOM_MAX 2147483647
#define NUM_OF_GPU_THREADS 1024

#ifndef FLT_MAX
#define FLT_MAX 3.40282347e+38
#endif

extern double wtime(void);

int find_nearest_point(float  *pt,          /* [nfeatures] */
                       int     nfeatures,
                       float **pts,         /* [npts][nfeatures] */
                       int     npts)
{
    int index, i;
    float min_dist=FLT_MAX;

    /* find the cluster center id with min distance to pt */
    for (i=0; i<npts; i++) {
        float dist;
        dist = euclid_dist_2(pt, pts[i], nfeatures);  /* no need square root */
        if (dist < min_dist) {
            min_dist = dist;
            index    = i;
        }
    }
    return(index);
}

/*----< euclid_dist_2() >----------------------------------------------------*/
/* multi-dimensional spatial Euclid distance square */
__inline
float euclid_dist_2(float *pt1,
                    float *pt2,
                    int    numdims)
{
    int i;
    float ans=0.0;

    for (i=0; i<numdims; i++)
        ans += (pt1[i]-pt2[i]) * (pt1[i]-pt2[i]);

    return(ans);
}

__device__ __inline float cuda_euclid_dist_2(float *pt1,
	float *pt2,
	int numdims)
{
	int i;
	float ans = 0.0;

	for (i = 0; i < numdims; i++)
		ans += (pt1[i] - pt2[i]) * (pt1[i] - pt2[i]);

	return (ans);
}

__device__  int cuda_find_nearest_point(float *pt, /* [nfeatures] */
	int nfeatures,
	float *pts, /* [npts][nfeatures] */
	int npts)
{
	int index, i;
	float min_dist = FLT_MAX;

	/* find the cluster center id with min distance to pt */
	for (i = 0; i < npts; i++)
	{
		float dist;
		dist = cuda_euclid_dist_2(pt, pts + i * nfeatures, nfeatures); /* no need square root */
		if (dist < min_dist)
		{
			min_dist = dist;
			index = i;
		}
	}
	return (index);
}

/*----< kmeans_clustering() >---------------------------------------------*/
float** kmeans_clustering(float **feature,    /* in: [npoints][nfeatures] */
                          int     nfeatures,
                          int     npoints,
                          int     nclusters,
                          float   threshold,
                          int    *membership) /* out: [npoints] */
{

    int      i, j, n=0, index, loop=0;
    int     *new_centers_len; /* [nclusters]: no. of points in each cluster */
    float    delta;
    float  **clusters;   /* out: [nclusters][nfeatures] */
    float  **new_centers;     /* [nclusters][nfeatures] */
  

    /* allocate space for returning variable clusters[] */
    clusters    = (float**) malloc(nclusters *             sizeof(float*));
    clusters[0] = (float*)  malloc(nclusters * nfeatures * sizeof(float));
    for (i=1; i<nclusters; i++)
        clusters[i] = clusters[i-1] + nfeatures;

    /* randomly pick cluster centers */
    for (i=0; i<nclusters; i++) {
        //n = (int)rand() % npoints;
        for (j=0; j<nfeatures; j++)
            clusters[i][j] = feature[n][j];
		n++;
    }

    for (i=0; i<npoints; i++)
		membership[i] = -1;

    /* need to initialize new_centers_len and new_centers[0] to all 0 */
    new_centers_len = (int*) calloc(nclusters, sizeof(int));

    new_centers    = (float**) malloc(nclusters *            sizeof(float*));
    new_centers[0] = (float*)  calloc(nclusters * nfeatures, sizeof(float));
    for (i=1; i<nclusters; i++)
        new_centers[i] = new_centers[i-1] + nfeatures;
 
  
    do {
		
        delta = 0.0;

        for (i=0; i<npoints; i++) {
	        /* find the index of nestest cluster centers */
	        index = find_nearest_point(feature[i], nfeatures, clusters, nclusters);
	        /* if membership changes, increase delta by 1 */
	        if (membership[i] != index) delta += 1.0;

	        /* assign the membership to object i */
	        membership[i] = index;

	        /* update new cluster centers : sum of objects located within */
	        new_centers_len[index]++;
	        for (j=0; j<nfeatures; j++)          
				new_centers[index][j] += feature[i][j];
        }
      

	/* replace old cluster centers with new_centers */
        for (i=0; i<nclusters; i++) {
            for (j=0; j<nfeatures; j++) {
                if (new_centers_len[i] > 0)
					clusters[i][j] = new_centers[i][j] / new_centers_len[i];
				new_centers[i][j] = 0.0;   /* set back to 0 */
			}
			new_centers_len[i] = 0;   /* set back to 0 */
		}
            
        //delta /= npoints;
    } while (delta > threshold);

  
    free(new_centers[0]);
    free(new_centers);
    free(new_centers_len);

    return clusters;
}




__global__ void updateNewCenters(float* cuda_delta_array, int* cuda_membership, float* cuda_features, int nfeatures, float* cuda_clusters, int nclusters, int* cuda_new_centers_len, float* cuda_new_centers) {

	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
	int local_id = threadIdx.x;

	float delta = 0;
	int membership = cuda_membership[global_id]; // global mem access

	int index = cuda_find_nearest_point(cuda_features + global_id * nfeatures, nfeatures, cuda_clusters, nclusters);

	if (membership != index) {
		delta = 1; // local delta every thread 

		// send to global delta array
	}

	cuda_delta_array[global_id] = delta;
	cuda_membership[global_id] = index;


	/*cuda_new_centers_len[index]++; // race condition
	for (j = 0; j < nfeatures; j++)
		new_centers[index][j] += feature[id][j];*/
}


/*----< kmeans_clustering_par() >---------------------------------------------*/
float **kmeans_clustering_par(float **feature, /* in: [npoints][nfeatures] */
	int nfeatures,
	int npoints,
	int nclusters,
	float threshold,
	int *membership) /* out: [npoints] */
{

	int i, j, n = 0, index, loop = 0;
	int *new_centers_len; /* [nclusters]: no. of points in each cluster */
	float delta;
	float **clusters;    /* out: [nclusters][nfeatures] */
	float **new_centers; /* [nclusters][nfeatures] */


	/* allocate space for returning variable clusters[] */
	clusters = (float **)malloc(nclusters * sizeof(float *));
	clusters[0] = (float *)malloc(nclusters * nfeatures * sizeof(float));
	for (i = 1; i < nclusters; i++)
		clusters[i] = clusters[i - 1] + nfeatures;

	/* randomly pick cluster centers */
	for (i = 0; i < nclusters; i++) {
		//n = (int)rand() % npoints;
		for (j = 0; j < nfeatures; j++)
			clusters[i][j] = feature[n][j];
		n++;
	}

	// INICIJALIZUJE CPU I KOPIRA U GLOBALNU MEMORIJU
	float * cuda_clusters;
	hipMalloc(&cuda_clusters, nclusters * nfeatures * sizeof(float));
	hipMemcpy(cuda_clusters, clusters[0], nclusters * nfeatures * sizeof(float), hipMemcpyHostToDevice);

	for (i = 0; i < npoints; i++)
		membership[i] = -1;
	int* cuda_membership;
	hipMalloc(&cuda_membership, npoints * sizeof(int));
	hipMemcpy(cuda_membership, membership, npoints * sizeof(int), hipMemcpyHostToDevice);

	// init cuda features
	float* cuda_features;
	hipMalloc(&cuda_features, npoints * nfeatures * sizeof(float));
	hipMemcpy(cuda_features, feature[0], npoints * nfeatures * sizeof(float), hipMemcpyHostToDevice);

	// INICIJALIZUJE CPU
	/* need to initialize new_centers_len and new_centers[0] to all 0 */
	new_centers_len = (int *)calloc(nclusters, sizeof(int));

	int* cuda_new_centers_len;
	hipMalloc(&cuda_new_centers_len, nclusters * sizeof(int));
	hipMemcpy(cuda_new_centers_len, new_centers_len, nclusters * sizeof(int), hipMemcpyHostToDevice);

	new_centers = (float **)malloc(nclusters * sizeof(float *));
	new_centers[0] = (float *)calloc(nclusters * nfeatures, sizeof(float));
	for (i = 1; i < nclusters; i++)
		new_centers[i] = new_centers[i - 1] + nfeatures;

	float * cuda_new_centers;
	hipMalloc(&cuda_new_centers, nclusters * nfeatures * sizeof(float));
	hipMemcpy(cuda_new_centers, new_centers[0], nclusters * nfeatures * sizeof(float), hipMemcpyHostToDevice);

	// create cuda_delta array
	float* cuda_delta_array;
	hipMalloc(&cuda_delta_array, npoints * sizeof(float));

	// SVAKA NIT KOPIRA SVOJ FEATURE[i] U SHARED MATRICU SHARED_FEATURES[1024][30]
	// SAMO NITI SA ID-EM 0..NCLUSTERS-1 KOPIRAJU CLUSTERS U SHARED_CLUSTERS

	dim3 gridDim((npoints + NUM_OF_GPU_THREADS - 1) / NUM_OF_GPU_THREADS);
	dim3 blockDim(NUM_OF_GPU_THREADS);

	//do
	//{
	// copy new_centers, new_centers_len, clusters		


	updateNewCenters <<< gridDim, blockDim >>> (cuda_delta_array, cuda_membership, cuda_features, nfeatures, cuda_clusters, nclusters, cuda_new_centers_len, cuda_new_centers);

	//	// KRAJ KERNELA
	//	// IMAMO REDUKOVANO NEW_CENTER I NEW_CENTERS_LEN KOJI SU U GLOBALNOJ CUDA MEMORIJI. KOPIRAMO IH U CPU MEMORIJU, IZRACUNAMO NOVO I NA OSNOVU
	//	// DELTA ODREDIMO DAL TREBANOVA ITERACIJA. AKO TREBA U CUDA PROSTOR KOPIRAMO CLUSTERS KOJI SMO IZRACUNALI
	//	// reduced new_centers and new_centers_len
	//	hipMemcpy(new_centers_len, cuda_new_centers_len, nclusters * sizeof(int), hipMemcpyDeviceToHost);
	//	hipMemcpy(new_centers[0], cuda_new_centers, nclusters * nfeatures * sizeof(float), hipMemcpyDeviceToHost);
	//	// CPU
	//	/* replace old cluster centers with new_centers */
	//	/*int xi, xj;
	//	for (xi = 0; xi < nclusters; xi++)
	//	{
	//		for (xj = 0; xj < nfeatures; xj++)
	//		{
	//			if (new_centers_len[xi] > 0)
	//				clusters[xi][xj] = new_centers[xi][xj] / new_centers_len[xi];
	//			new_centers[xi][xj] = 0.0; /* set back to 0 */
	//		}
	//		/*new_centers_len[xi] = 0; /* set back to 0 */
	//	//}
	//	// delta reduction
	//	// deltaReduction<<< gridDim, blockDim >>>();
	//	//delta /= npoints;
	//} while (delta > threshold);*/
	//free(new_centers[0]);
	//free(new_centers);
	//free(new_centers_len);
	//return clusters;*/
}