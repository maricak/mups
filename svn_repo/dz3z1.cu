
#include "hip/hip_runtime.h"


#include <stdio.h>

#include <string>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <malloc.h>
#include <vector>
#include <iostream>
#include <fstream>

#include <functional>
#include <algorithm>

#include <ctime>

#define ACCURACY 0.01
#define NUM_OF_GPU_THREADS 2
#define BLOCK_SIZE 32
#define NUM_OF_GPU_BLOCKS 4

#define MAX 3
#define MIN 0


//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//	int i = threadIdx.x;
//	c[i] = a[i] + b[i];
//}
//
//int main()
//{
//	const int arraySize = 5;
//	const int a[arraySize] = { 1, 2, 3, 4, 5 };
//	const int b[arraySize] = { 10, 20, 30, 40, 50 };
//	int c[arraySize] = { 0 };
//
//	// Add vectors in parallel.
//	cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "addWithCuda failed!");
//		return 1;
//	}
//
//	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//		c[0], c[1], c[2], c[3], c[4]);
//
//	// cudaDeviceReset must be called before exiting in order for profiling and
//	// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	cudaStatus = cudaDeviceReset();
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaDeviceReset failed!");
//		return 1;
//	}
//
//	return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//	int *dev_a = 0;
//	int *dev_b = 0;
//	int *dev_c = 0;
//	cudaError_t cudaStatus;
//
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	cudaStatus = cudaSetDevice(0);
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//		goto Error;
//	}
//
//	// Allocate GPU buffers for three vectors (two input, one output)    .
//	cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMalloc failed!");
//		goto Error;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMemcpy failed!");
//		goto Error;
//	}
//
//	cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMemcpy failed!");
//		goto Error;
//	}
//
//	// Launch a kernel on the GPU with one thread for each element.
//	addKernel << <1, size >> > (dev_c, dev_a, dev_b);
//
//	// Check for any errors launching the kernel
//	cudaStatus = cudaGetLastError();
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// cudaDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = cudaDeviceSynchronize();
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		goto Error;
//	}
//
//	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMemcpy failed!");
//		goto Error;
//	}
//
//Error:
//	cudaFree(dev_c);
//	cudaFree(dev_a);
//	cudaFree(dev_b);
//
//	return cudaStatus;
//}

void checkError(hipError_t err, int line) {
	if (hipSuccess != err) {
		std::cerr << "Error " << hipGetErrorName(err) << " happenend: " << hipGetErrorString(err) << " at line " << line << std::endl;
		exit(-1);
	}
}

//void matrixInit(float * matrix, int size) {
//	for (int i = 0; i < size; ++i) {
//		matrix[i] = rand() % (MAX - MIN + 1) + MIN;
//	}
//}

__global__ void sgemmKernel(float * cudaA, float * cudaB, float * cudaC, int m, int n, int k, float alpha, float beta) {

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	__shared__ float A[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float B[BLOCK_SIZE][BLOCK_SIZE];

	int row = bx * BLOCK_SIZE + tx;
	int col = by * BLOCK_SIZE + ty;

	float sum = 0;

	for (int i = 0; i < gridDim.y; i++) {

		int a_col = i * BLOCK_SIZE + ty;
		int a_row = row;

		if (a_row < m && a_col < k) {
			A[tx][ty] = cudaA[a_row + a_col * m];
		}

		int b_col = col;
		int b_row = i * BLOCK_SIZE + tx;

		if (b_row < k && b_col < n){
			B[tx][ty] = cudaB[b_row * n + b_col];
		}

		__syncthreads();

		if (row < m && col < n) {

			int j_end = ((i + 1) * BLOCK_SIZE < k) ? BLOCK_SIZE : k - i * BLOCK_SIZE;


			for (int j = 0; j < j_end; j++) {
				sum += A[tx][j] * B[j][ty];
			}
		}
		__syncthreads();
	}
	if (row < m && col < n) {
		cudaC[row + col * m] = 0;
		cudaC[row + col * m] = cudaC[row + col * m] * beta + sum * alpha;
	}
}
bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float>&v)
{
	std::cerr << "Opening file:" << fn << std::endl;
	std::fstream f(fn, std::fstream::in);
	if (!f.good()) {
		return false;
	}

	// Read # of rows and cols
	f >> nr_row;
	f >> nr_col;

	float data;
	std::cerr << "Matrix dimension: " << nr_row << "x" << nr_col << std::endl;
	while (f.good()) {
		f >> data;
		v.push_back(data);
	}
	v.pop_back(); // remove the duplicated last element

	return true;
}

bool writeColMajorMatrixFile(const char *fn, int nr_row, int nr_col, std::vector<float>&v)
{
	std::cerr << "Opening file:" << fn << " for write." << std::endl;
	std::fstream f(fn, std::fstream::out);
	if (!f.good()) {
		return false;
	}

	// Read # of rows and cols
	f << nr_row << " " << nr_col << " ";

	std::cerr << "Matrix dimension: " << nr_row << "x" << nr_col << std::endl;
	for (int i = 0; i < v.size(); ++i) {
		f << v[i] << ' ';
	}
	f << "\n";
	return true;

}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
	if ((transa != 'N') && (transa != 'n')) {
		std::cerr << "unsupported value of 'transa' in regtileSgemm()" << std::endl;
		return;
	}

	if ((transb != 'T') && (transb != 't')) {
		std::cerr << "unsupported value of 'transb' in regtileSgemm()" << std::endl;
		return;
	}

	for (int mm = 0; mm < m; ++mm) {
		for (int nn = 0; nn < n; ++nn) {
			float c = 0.0f;
			for (int i = 0; i < k; ++i) {
				float a = A[mm + i * lda];
				float b = B[nn + i * ldb];
				c += a * b;
			}
			C[mm + nn * ldc] = C[mm + nn * ldc] * beta + alpha * c;
		}
	}
}

void basicSgemm_par(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
	float *cudaA, *cudaB, *cudaC;
	int sizeA = m * k * sizeof(float), sizeB = k * n * sizeof(float), sizeC = m * n * sizeof(float);

	if ((transa != 'N') && (transa != 'n')) {
		std::cerr << "unsupported value of 'transa' in regtileSgemm()" << std::endl;
		return;
	}

	if ((transb != 'T') && (transb != 't')) {
		std::cerr << "unsupported value of 'transb' in regtileSgemm()" << std::endl;
		return;
	}

	checkError(hipMalloc(&cudaA, sizeA), __LINE__);
	checkError(hipMemcpy(cudaA, A, sizeA, hipMemcpyHostToDevice), __LINE__);

	checkError(hipMalloc(&cudaB, sizeB), __LINE__);
	checkError(hipMemcpy(cudaB, B, sizeB, hipMemcpyHostToDevice), __LINE__);

	checkError(hipMalloc(&cudaC, sizeC), __LINE__);

	dim3 dimGrid((m + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	sgemmKernel <<< dimGrid, dimBlock >>> (cudaA, cudaB, cudaC, m, n, k, alpha, beta);

	checkError(hipMemcpy(C, cudaC, sizeC, hipMemcpyDeviceToHost), __LINE__);

	checkError(hipFree(cudaA), __LINE__);
	checkError(hipFree(cudaB), __LINE__);
	checkError(hipFree(cudaC), __LINE__);
}

int main(int argc, char *argv[]) {

	int matArow, matAcol;
	int matBrow, matBcol;

	std::vector<float> matA;
	std::vector<float> matBT;

	//int m, n, k;

	float timeSeq = 0.0f, timePar = 0.0f;

	hipEvent_t start = hipEvent_t();
	checkError(hipEventCreate(&start), __LINE__);
	hipEvent_t stop = hipEvent_t();
	checkError(hipEventCreate(&stop), __LINE__);

	if (argc != 4)
	{
		fprintf(stderr, "Expecting three input filenames\n");
		exit(-1);
	}


	/* Read in data */
	// load A
	readColMajorMatrixFile(argv[1], matArow, matAcol, matA);

	// load B^T
	readColMajorMatrixFile(argv[2], matBcol, matBrow, matBT);

	/*m = std::stoi(argv[1]);
	n = std::stoi(argv[2]);
	k = std::stoi(argv[3]);
	// A - m*k
	// B - k*n
	// C - m*n

	matArow = m;
	matAcol = k;
	matBrow = k;
	matBcol = n;
	// allocate space for A and B
	//matrixInit(&matA.front(), m*k);
	//matrixInit(&matBT.front(), k*n);*/

	// allocate space for C and D
	std::vector<float> matC(matArow * matBcol);
	std::vector<float> matD(matArow * matBcol);


	//clock_t begin = clock();
	hipEventRecord(start, 0);
	// Use standard sgemm interface
	basicSgemm('N', 'T', matArow, matBcol, matAcol, 1.0f, &matA.front(), matArow, &matBT.front(), matBcol, 0.0f, &matC.front(), matArow);
	clock_t end = clock();
	//timeSeq = float(end - begin) / CLOCKS_PER_SEC;
	checkError(hipEventRecord(stop, 0), __LINE__);
	checkError(hipEventSynchronize(stop), __LINE__);
	checkError(hipEventElapsedTime(&timeSeq, start, stop), __LINE__);
	timeSeq /= 1000;

	hipEventRecord(start, 0);
	// Use parallel sgemm interface
	basicSgemm_par('N', 'T', matArow, matBcol, matAcol, 1.0f, &matA.front(), matArow, &matBT.front(), matBcol, 0.0f, &matD.front(), matArow);
	checkError(hipEventRecord(stop, 0), __LINE__);
	checkError(hipEventSynchronize(stop), __LINE__);
	checkError(hipEventElapsedTime(&timePar, start, stop), __LINE__);
	timePar /= 1000;

	checkError(hipEventDestroy(start), __LINE__);
	checkError(hipEventDestroy(stop), __LINE__);

	writeColMajorMatrixFile(argv[3], matArow, matBcol, matC);

	std::function<bool(double, double)> comparator = [](double left, double right) {
		// Lambda function to compare 2 doubles with ACCURACY
		return fabs(left - right) < ACCURACY;
	};

	std::cerr << "********************DZ3Z1**********************" << std::endl;
	std::cerr << "Elapsed time - SEQ: " << timeSeq << "." << std::endl;
	std::cerr << "Elapsed time - PAR: " << timePar << "." << std::endl;
	std::cerr << (std::equal(matC.begin(), matC.end(), matD.begin(), comparator) ? "TEST PASSED" : "TEST FAILED") << std::endl;
	std::cerr << "***********************************************" << std::endl;

	return 0;
}

